/*
#_*_coding : UTF-8_*_
# Code writer: Weiguang.Zhao
# Writing time: 2021/9/24  下午10:56
# File Name: KernelNesting
# IDE: CLion
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void  sub_kernel( )
{
    int th_index = blockIdx.x*blockDim.x + threadIdx.x;
    printf("-------> sub_kernel thread number: %d \n", th_index);
}

__global__ void  kernel( )
{
    int th_index = blockIdx.x*blockDim.x + threadIdx.x;
    printf("-------> kernel thread number: %d \n", th_index);
//    sub_kernel<<<2,2>>>();
    kernel<<<2,2>>>();
}

int main(void )
{
    kernel<<<2,2>>>();
    hipDeviceReset();
    return 0;
}