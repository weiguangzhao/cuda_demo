#include "hip/hip_runtime.h"
/*
#_*_coding : UTF-8_*_
# Code writer: Weiguang.Zhao
# Writing time: 2021/9/23  下午2:29
# File Name: square
# IDE: CLion
 */

# include "square.h"
#include <stdio.h>


__global__ void square(float * d_out, float * d_in){
    int idx = threadIdx.x;
    float f =d_in[idx];
    d_out[idx] = f*f;
};

void square_test(){

    // 定义数组长度和位数
    int array_size = 64;
    int array_bytes = array_size * sizeof(float);

    // 产生数组
    float h_in[array_size];  //输入数组
    for (int  i=0; i< array_size; i++){
        h_in[i] = float(i);
    }
    float h_out[array_size]; //存储结果

    // 定义GPU内存指针
    float * d_in;
    float * d_out;

    // 分配GPU内存
    hipMalloc((void **) &d_in, array_bytes);
    hipMalloc((void **) &d_out, array_bytes);

    // 把CPU数据搬到GPU上
    hipMemcpy(d_in, h_in, array_bytes, hipMemcpyHostToDevice);

    // 运行cuda内核开始计算
    square<<<1, array_size>>>(d_out, d_in);

    // 把运算结果搬回CPU
    hipMemcpy(h_out, d_out, array_bytes, hipMemcpyDeviceToHost);

    // 打印结果
    for (int i =0; i< array_size; i++){
        printf("%f \n", h_out[i]);
    }

    // 释放GPU内存
    hipFree(d_in);
    hipFree(d_out);
}