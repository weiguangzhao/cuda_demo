#include "hip/hip_runtime.h"
/*
#_*_coding : UTF-8_*_
# Code writer: Weiguang.Zhao
# Writing time: 2021/10/3  下午7:09
# File Name: AdjacentPoint
# IDE: CLion
*/
#include <stdio.h>
#include <stdlib.h>

#include "cuda_config.h"
#include "AdjacentPoint.h"

// https://stackoverflow.com/a/14038590
#define CUDA_ERR_CHK(code) { cuda_err_chk((code), __FILE__, __LINE__); }
inline void cuda_err_chk(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "\tCUDA ERROR: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void checkNearPoints_cuda(int *point_num_d, float *xyz_d, float *eps_d, int *ptsCnt_d){
    int th_index = blockIdx.x*blockDim.x + threadIdx.x;
    if (th_index >= *point_num_d) return ;

    ptsCnt_d[th_index] = 0;  // the number of adjacent points
    float o_x = xyz_d[th_index * 3 + 0];
    float o_y = xyz_d[th_index * 3 + 1];
    float o_z = xyz_d[th_index * 3 + 2];

    for (int k =0; k< *point_num_d; k++){
        if(th_index==k) continue;

        float k_x = xyz_d[k * 3 + 0];
        float k_y = xyz_d[k * 3 + 1];
        float k_z = xyz_d[k * 3 + 2];
        float l2 = sqrt((k_x-o_x)*(k_x-o_x)+(k_y-o_y)*(k_y-o_y)+(k_z-o_z)*(k_z-o_z));
        if (l2 <= *eps_d) {
            ptsCnt_d[th_index]= ptsCnt_d[th_index] + 1;
        }
    }
}

void checkNearPoints(const int point_num, float xyz[], const float radius){
    dim3 blocks(DIVUP(point_num, THREADS_PER_BLOCK));
    dim3 threads(THREADS_PER_BLOCK);


    int * ptsCnt_h;
    ptsCnt_h = (int *)malloc(point_num * sizeof(int));

    // define gpu variable
    int * point_num_d;
    float * xyz_d;
    float * radius_d;
    int * ptsCnt_d; //mark the number of adjacent points


    //generate gpu ram
    CUDA_ERR_CHK( hipMalloc((void **) &point_num_d, sizeof(int)));
    CUDA_ERR_CHK(hipMalloc((void **) &xyz_d, 3*point_num*sizeof(float)));
    CUDA_ERR_CHK(hipMalloc((void **) &radius_d, sizeof(float)));
    CUDA_ERR_CHK(hipMalloc((void **) &ptsCnt_d, point_num*sizeof(int)));


    // copy host to device
    CUDA_ERR_CHK(hipMemcpy(point_num_d, &point_num, sizeof(int), hipMemcpyHostToDevice));
    CUDA_ERR_CHK(hipMemcpy(xyz_d, xyz, 3*point_num*sizeof(float), hipMemcpyHostToDevice));
    CUDA_ERR_CHK(hipMemcpy(radius_d, &radius, sizeof(float), hipMemcpyHostToDevice));

    // start device kernel
    checkNearPoints_cuda<<<blocks, threads>>>(point_num_d, xyz_d, radius_d, ptsCnt_d);

    // copy device to host
    CUDA_ERR_CHK(hipMemcpy(ptsCnt_h, ptsCnt_d, point_num*sizeof(float), hipMemcpyDeviceToHost));

    // release gpu ram
    CUDA_ERR_CHK(hipFree(point_num_d));
    CUDA_ERR_CHK(hipFree(xyz_d));
    CUDA_ERR_CHK(hipFree(radius_d));
    CUDA_ERR_CHK(hipFree(ptsCnt_d));

    for (int i =0; i< point_num; i++){
        printf("index: %d adjacent point number: %d \n",i, ptsCnt_h[i]);
    }

}